#include "hip/hip_runtime.h"
// make setup; make gpu_join; ./bin/gpu/join > out
// Ensure printing of CUDA runtime errors to console
#define HIPCUB_STDERR

#include <iostream>
#include <stdio.h>
#include <hiprand.h>

#include <hip/hip_runtime.h>
#include <cub/util_allocator.cuh>
#include <cub/device/device_scan.cuh>
#include <cub/device/device_radix_sort.cuh>
#include "cub/test/test_util.h"

#include "utils/generator.h"
#include "utils/gpu_utils.h"

using namespace std;
using namespace hipcub;

#define DEBUG 1
#define NGPU 2
#define PRINT 1

__device__ __forceinline__
int HASH(const int key, const int num_slots) {
  return key & (num_slots - 1);
}

__forceinline__
int HHASH(const int key, const int num_slots) {
  return key & (num_slots - 1);
}

void partition_dev(int* h_key, int* h_val, int** key_partitions, int** val_partitions, int num_slots, int count[NGPU]) {

  for (int i = 0; i < num_slots; i ++) {
    int key = h_key[i];
    int hash = (HHASH(key, num_slots) % NGPU);

    key_partitions[hash][count[hash]] = key;
    val_partitions[hash][count[hash]] = h_val[i];

    count[hash]++;
  }
}

__global__
void build_hashtable_dev(int *d_dim_key, int *d_dim_val, int num_tuples, int *hash_table, int num_slots) {
  int offset = blockIdx.x * blockDim.x + threadIdx.x;

  int key = d_dim_key[offset];
  int val = d_dim_val[offset];
  int hash = HASH(key, num_slots);

  if (offset < num_tuples) {
    hash_table[hash << 1] = key;
    hash_table[(hash << 1) + 1] = val;
    if (PRINT) printf("Wrote %d to position %d in hashtable\n", key, hash << 1);
  }
}

__global__
void probe_hashtable_dev(int *d_fact_fkey, int *d_fact_val, int num_tuples, int *hash_table, int num_slots, unsigned long long *res) {
  int offset = blockIdx.x*blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;

  unsigned long long checksum = 0;

  for (int i = offset; i < num_tuples; i += stride) {
    int key = d_fact_fkey[i];
    if (PRINT) printf("Fact key at %d is %d\n", i, key);
    int val = d_fact_val[i];
    int hash = HASH(key, num_slots);

    int2 slot = reinterpret_cast<int2*>(hash_table)[hash];
    if (PRINT) printf("Key at hash %d is %d\n", hash, slot.x);
    if (slot.x == key) {
      if (PRINT) printf("%d matches! Adding %d and %d\n", key, slot.y, val);
      checksum += slot.y + val;
    }
  }

  atomicAdd(res, checksum);
}

struct TimeKeeper {
  float time_build;
  float time_probe;
  float time_extra;
  float time_total;
};

static int num_runs = 0;
static unsigned long long* h_res = 0;

TimeKeeper hashJoin(int* h_dim_key, int* h_dim_val, int* h_fact_fkey, int* h_fact_val, int num_dim, int num_fact, CachingDeviceAllocator&  g_allocator) {
  SETUP_TIMING();

  // Partition
  int** h_dim_key_partitions = new int*[NGPU];
  for(int i = 0; i < NGPU; ++i) h_dim_key_partitions[i] = new int[num_dim];
  int** h_dim_val_partitions = new int*[NGPU];
  for(int i = 0; i < NGPU; ++i) h_dim_val_partitions[i] = new int[num_dim];
  int** h_fact_key_partitions = new int*[NGPU];
  for(int i = 0; i < NGPU; ++i) h_fact_key_partitions[i] = new int[num_fact];
  int** h_fact_val_partitions = new int*[NGPU];
  for(int i = 0; i < NGPU; ++i) h_fact_val_partitions[i] = new int[num_fact];
  int* h_dim_count = new int[NGPU]; 
  int* h_fact_count = new int[NGPU];

  memset(h_dim_count, 0, sizeof(int)*NGPU); 
  memset(h_fact_count, 0, sizeof(int)*NGPU); 

  if (PRINT) printf("Partitioning dim...\n");
  partition_dev(h_dim_key, h_dim_val, h_dim_key_partitions, h_dim_val_partitions, num_dim, h_dim_count); 
  for (int i = 0; i < 2; i++) {
    for (int j = 0; j < h_dim_count[i]; j++) {
      if (PRINT) printf("%d:%d ", h_dim_key_partitions[i][j], h_dim_val_partitions[i][j]);
    }
    if (PRINT) printf("\n");
  } 

  if (PRINT) printf("Partitioning fact...\n");
  partition_dev(h_fact_fkey, h_fact_val, h_fact_key_partitions, h_fact_val_partitions, num_fact, h_fact_count);
  for (int i = 0; i < NGPU; i++) {
    for (int j = 0; j < h_fact_count[i]; j++) {
      if (PRINT)  printf("%d:%d ", h_fact_key_partitions[i][j], h_fact_val_partitions[i][j]);
    }
    if (PRINT) printf("\n");
  } 

  // Build hashtable (TODO: N hashtables) 
  int* hash_table_0;
  int* hash_table_1; 
  unsigned long long* res;
  int num_slots = num_dim;
  float time_build, time_probe, time_memset, time_memset2;

  int (*d_dim_key_partitions)[NGPU];
  int (*d_dim_val_partitions)[NGPU]; 
  int (*d_fact_key_partitions)[NGPU];
  int (*d_fact_val_partitions)[NGPU]; 

  ALLOCATE(hash_table_0, sizeof(int) * 2 * num_dim);
  ALLOCATE(hash_table_1, sizeof(int) * 2 * num_dim);
  ALLOCATE(res, sizeof(long long));

  ALLOCATE(d_dim_key_partitions, sizeof(int) * NGPU * num_dim);
  ALLOCATE(d_dim_val_partitions, sizeof(int) * NGPU * num_dim); 
  ALLOCATE(d_fact_key_partitions, sizeof(int) * NGPU * num_fact); 
  ALLOCATE(d_fact_val_partitions, sizeof(int) * NGPU * num_fact); 

  TIME_FUNC(hipMemset(d_dim_key_partitions, 0, NGPU * num_dim * sizeof(int)), time_memset);
  TIME_FUNC(hipMemset(d_dim_val_partitions, 0, NGPU * num_dim * sizeof(int)), time_memset); 

  for (int i = 0; i < NGPU; i++) {
    CubDebugExit(hipMemcpy(d_dim_key_partitions[i], h_dim_key_partitions[i], sizeof(int) * h_dim_count[i],  hipMemcpyHostToDevice));
    CubDebugExit(hipMemcpy(d_dim_val_partitions[i], h_dim_val_partitions[i], sizeof(int) * h_dim_count[i], hipMemcpyHostToDevice));

    printf("%d round: Copying 4 values to pointer %d = %d\n", i, d_fact_key_partitions + i*num_fact*sizeof(int), d_fact_key_partitions[i]);

    CubDebugExit(hipMemcpy(d_fact_key_partitions + i * num_fact * sizeof(int), h_fact_key_partitions[i], sizeof(int) * h_fact_count[i], hipMemcpyHostToDevice));
    CubDebugExit(hipMemcpy(d_fact_val_partitions + i * num_fact * sizeof(int), h_fact_val_partitions[i], sizeof(int) * h_fact_count[i], hipMemcpyHostToDevice));
  }

  TIME_FUNC(hipMemset(hash_table_0, 0, 2 * num_slots * sizeof(int)), time_memset);
  TIME_FUNC(hipMemset(hash_table_1, 0, 2 * num_slots * sizeof(int)), time_memset); 
  TIME_FUNC(hipMemset(res, 0, sizeof(long long)), time_memset2);

  if (PRINT) printf("\nBuilding hashtable 0...\n");
  // num_dim/128
  TIME_FUNC((build_hashtable_dev<<<128, 128>>>(d_dim_key_partitions[0], d_dim_val_partitions[0], h_dim_count[0], hash_table_0, num_slots)), time_build);
  hipDeviceSynchronize(); 

  if (PRINT) printf("Building hashtable 1...\n");
  TIME_FUNC((build_hashtable_dev<<<128, 128>>>(d_dim_key_partitions[1], d_dim_val_partitions[1], h_dim_count[1], hash_table_1, num_slots)), time_build);
  hipDeviceSynchronize(); 

  // Probe hashtable

  if (PRINT) printf("\nProbing hashtable 0...\n");
  TIME_FUNC((probe_hashtable_dev<<<192, 256>>>(d_fact_key_partitions[0], d_fact_val_partitions[0], h_fact_count[0], hash_table_0, num_slots, res)), time_probe);
  hipDeviceSynchronize(); 

  if (PRINT) printf("Probing hashtable 1...\n");
  TIME_FUNC((probe_hashtable_dev<<<192, 256>>>(d_fact_key_partitions[1], d_fact_val_partitions[1], h_fact_count[1], hash_table_1, num_slots, res)), time_probe);
  hipDeviceSynchronize(); 

#if DEBUG
  cout << "{" << "\"time_memset\":" << time_memset
    << ",\"time_build\"" << time_build
    << ",\"time_probe\":" << time_probe << "}" << endl;
#endif

  num_runs += 1;
  if (num_runs == 3) {
    h_res = new unsigned long long[1];
    CubDebugExit(hipMemcpy(h_res, res, sizeof(long long), hipMemcpyDeviceToHost));
    cout << h_res[0] << endl;
  }

  CLEANUP(hash_table_0);
  CLEANUP(hash_table_1); 
  CLEANUP(res);
  CLEANUP(d_dim_key_partitions);
  CLEANUP(d_dim_val_partitions); 
  CLEANUP(d_fact_key_partitions);
  CLEANUP(d_fact_val_partitions); 

  TimeKeeper t = {time_build, time_probe, time_memset, time_build + time_probe + time_memset};
  return t;
}

void RunHashJoinCPU(int *dim_key, int* dim_val, int* fact_fkey, int* fact_val, int* hash_table, long long* res, int num_dim, int num_fact, int num_slots) {
  for (int i = 0; i < num_dim; i++) {
    int key = dim_key[i];
    int val = dim_val[i];

    int hash = key & (num_slots - 1);

    hash_table[hash << 1] = key;
    hash_table[(hash << 1) + 1] = val;
  }

  for (int i = 0; i < num_fact; i++) {
    int key = fact_fkey[i];
    int val = fact_val[i];

    int hash = key & (num_slots - 1);

    if (hash_table[hash << 1] == key) {
      printf("%d matches! Adding %d and %d\n", key, hash_table[(hash << 1) + 1], val);
      *res = *res + (hash_table[(hash << 1) + 1] + val);
      printf("res is now %d\n", *res);
    } 
  }
}

//---------------------------------------------------------------------
// Globals, constants and typedefs
//---------------------------------------------------------------------

bool                    g_verbose = false;  // Whether to display input/output to console
CachingDeviceAllocator  g_allocator(true);  // Caching allocator for device memory


#define CLEANUP(vec) if(vec)CubDebugExit(g_allocator.DeviceFree(vec))

//---------------------------------------------------------------------
// Main
//---------------------------------------------------------------------
int main(int argc, char** argv)
{
  int num_fact           = 8; // 256 * 1 << 20 , 1 << 28
  int num_dim            = 4; // 16 * 1 << 20 , 1 << 16
  int num_trials         = 3;

  // Initialize command line
  CommandLineArgs args(argc, argv);
  args.GetCmdLineArgument("n", num_fact);
  args.GetCmdLineArgument("d", num_dim);
  args.GetCmdLineArgument("t", num_trials);

  // Print usage
  if (args.CheckCmdLineFlag("help"))
  {
    printf("%s "
        "[--n=<num fact>] "
        "[--d=<num dim>] "
        "[--t=<num trials>] "
        "[--device=<device-id>] "
        "[--v] "
        "\n", argv[0]);
    exit(0);
  }

  int log2 = 0;
  int num_dim_dup = num_dim >> 1;
  while (num_dim_dup) {
    num_dim_dup >>= 1;
    log2 += 1;
  }

  // Initialize device
  CubDebugExit(args.DeviceInit());

  int *h_dim_key = NULL;
  int *h_dim_val = NULL;
  int *h_fact_fkey = NULL;
  int *h_fact_val = NULL;

  create_relation_pk(h_dim_key, h_dim_val, num_dim);
  create_relation_fk(h_fact_fkey, h_fact_val, num_fact, num_dim);

  if (PRINT) {
    cout << "DIM TABLE:" << endl;
    for (int i = 0; i < num_dim; i++) cout << h_dim_key[i] << "..." << h_dim_val[i] << endl;
    cout << endl;

    cout << "FACT TABLE:" << endl;
    for (int i = 0; i < num_fact; i++) cout << h_fact_fkey[i] << "..." << h_fact_val[i] << endl;
    cout << endl;
  }

  for (int j = 0; j < num_trials; j++) {
    cout << "TRIAL " << j << endl;
    TimeKeeper t = hashJoin(h_dim_key, h_dim_val, h_fact_fkey, h_fact_val, num_dim, num_fact, g_allocator);
    cout<< "{"
      << "\"num_dim\":" << num_dim
      << ",\"num_fact\":" << num_fact
      << ",\"radix\":" << 0
      << ",\"time_partition_build\":" << 0
      << ",\"time_partition_probe\":" << 0
      << ",\"time_partition_total\":" << 0
      << ",\"time_build\":" << t.time_build
      << ",\"time_probe\":" << t.time_probe
      << ",\"time_extra\":" << t.time_extra
      << ",\"time_join_total\":" << t.time_total
      << "}" << endl;
    cout << endl;
  }

  // Checking against hash join on CPU
  int num_slots = num_dim;
  int *hash_table = new int[num_slots * 2];
  long long check_res = 0;
  RunHashJoinCPU(h_dim_key, h_dim_val, h_fact_fkey, h_fact_val, hash_table, &check_res, num_dim, num_fact, num_slots);
  cout << "CPU answer: " << check_res << endl; 

  return 0;
}

