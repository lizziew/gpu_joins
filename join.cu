#include "hip/hip_runtime.h"
// make setup; make gpu_join; ./bin/gpu/join > out
// Ensure printing of CUDA runtime errors to console
#define HIPCUB_STDERR

#include <iostream>
#include <stdio.h>
#include <hiprand.h>

#include <hip/hip_runtime.h>
#include <cub/util_allocator.cuh>
#include <cub/device/device_scan.cuh>
#include <cub/device/device_radix_sort.cuh>
#include "cub/test/test_util.h"

#include "utils/generator.h"
#include "utils/gpu_utils.h"

using namespace std;
using namespace hipcub;

#define DEBUG 1

__device__ __forceinline__
int HASH(const int key, const int num_slots) {
  return key & (num_slots - 1);
}

__global__
void build_hashtable_dev(int *d_dim_key, int *d_dim_val, int num_tuples, int *hash_table, int num_slots) {
  int offset = blockIdx.x * blockDim.x + threadIdx.x;
  int key = d_dim_key[offset];
  int val = d_dim_val[offset];
  int hash = HASH(key, num_slots);

  hash_table[hash << 1] = key;
  hash_table[(hash << 1) + 1] = val;
  // printf("key and value are %d %d\n", key, val);
}

__global__
void probe_hashtable_dev(int *d_fact_fkey, int *d_fact_val, int num_tuples, int *hash_table, int num_slots, unsigned long long *res) {
  int offset = blockIdx.x*blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;

  unsigned long long checksum = 0;

  for (int i = offset; i < num_tuples; i += stride) {
    int key = d_fact_fkey[i];
    int val = d_fact_val[i];
    int hash = HASH(key, num_slots);

    int2 slot = reinterpret_cast<int2*>(hash_table)[hash];
    // printf("Hmmmm %d %d\n", slot.x, key);
    if (slot.x == key) {
      // printf("%d %d\n", slot.x, key);
      checksum += slot.y + val;
    }
  }

  atomicAdd(res, checksum);
}

struct TimeKeeper {
  float time_build;
  float time_probe;
  float time_extra;
  float time_total;
};

static int num_runs = 0;
static unsigned long long* h_res = 0;

TimeKeeper hashJoin(int* d_dim_key, int* d_dim_val, int* d_fact_fkey, int* d_fact_val, int num_dim, int num_fact, CachingDeviceAllocator&  g_allocator) {
  SETUP_TIMING();

  int* hash_table = NULL;
  unsigned long long* res;
  int num_slots = num_dim;
  float time_build, time_probe, time_memset, time_memset2;

  ALLOCATE(hash_table, sizeof(int) * 2 * num_dim);
  ALLOCATE(res, sizeof(long long));

  TIME_FUNC(hipMemset(hash_table, 0, num_slots * sizeof(int) * 2), time_memset);
  TIME_FUNC(hipMemset(res, 0, sizeof(long long)), time_memset2);

  TIME_FUNC((build_hashtable_dev<<<num_dim/128, 128>>>(d_dim_key, d_dim_val, num_dim, hash_table, num_slots)), time_build);

  hipDeviceSynchronize(); 

  TIME_FUNC((probe_hashtable_dev<<<192, 256>>>(d_fact_fkey, d_fact_val, num_fact, hash_table, num_slots, res)), time_probe);
  hipDeviceSynchronize(); 

#if DEBUG
  cout << "{" << "\"time_memset\":" << time_memset
    << ",\"time_build\"" << time_build
    << ",\"time_probe\":" << time_probe << "}" << endl;
#endif

  num_runs += 1;
  if (num_runs == 3) {
    h_res = new unsigned long long[1];
    CubDebugExit(hipMemcpy(h_res, res, sizeof(long long), hipMemcpyDeviceToHost));
    cout << h_res[0] << endl;
  }

  CLEANUP(hash_table);
  CLEANUP(res);

  TimeKeeper t = {time_build, time_probe, time_memset, time_build + time_probe + time_memset};
  return t;
}

void RunHashJoinCPU(int *dim_key, int* dim_val, int* fact_fkey, int* fact_val, int* hash_table, long long* res, int num_dim, int num_fact, int num_slots) {
  for (int i = 0; i < num_dim; i++) {
    int key = dim_key[i];
    int val = dim_val[i];

    int hash = key & (num_slots - 1);

    hash_table[hash << 1] = key;
    hash_table[(hash << 1) + 1] = val;
  }

  for (int i = 0; i < num_fact; i++) {
    int key = fact_fkey[i];
    int val = fact_val[i];

    int hash = key & (num_slots - 1);

    if (hash_table[hash << 1] == key) {
      *res = *res + (hash_table[(hash << 1) + 1] + val);
    } 
  }
}

//---------------------------------------------------------------------
// Globals, constants and typedefs
//---------------------------------------------------------------------

bool                    g_verbose = false;  // Whether to display input/output to console
CachingDeviceAllocator  g_allocator(true);  // Caching allocator for device memory


#define CLEANUP(vec) if(vec)CubDebugExit(g_allocator.DeviceFree(vec))

//---------------------------------------------------------------------
// Main
//---------------------------------------------------------------------
int main(int argc, char** argv)
{
  int num_fact           = 256 * 1 << 20; // 256 * 1 << 20 , 1 << 28
  int num_dim            = 16 * 1 << 20; // 16 * 1 << 20 , 1 << 16
  int num_trials         = 3;

  // Initialize command line
  CommandLineArgs args(argc, argv);
  args.GetCmdLineArgument("n", num_fact);
  args.GetCmdLineArgument("d", num_dim);
  args.GetCmdLineArgument("t", num_trials);

  // Print usage
  if (args.CheckCmdLineFlag("help"))
  {
    printf("%s "
        "[--n=<num fact>] "
        "[--d=<num dim>] "
        "[--t=<num trials>] "
        "[--device=<device-id>] "
        "[--v] "
        "\n", argv[0]);
    exit(0);
  }

  int log2 = 0;
  int num_dim_dup = num_dim >> 1;
  while (num_dim_dup) {
    num_dim_dup >>= 1;
    log2 += 1;
  }

  // Initialize device
  CubDebugExit(args.DeviceInit());

  // Allocate problem device arrays
  int *d_dim_key = NULL;
  int *d_dim_val = NULL;
  int *d_fact_fkey = NULL;
  int *d_fact_val = NULL;
  CubDebugExit(g_allocator.DeviceAllocate((void**)&d_dim_key, sizeof(int) * num_dim));
  CubDebugExit(g_allocator.DeviceAllocate((void**)&d_dim_val, sizeof(int) * num_dim));
  CubDebugExit(g_allocator.DeviceAllocate((void**)&d_fact_fkey, sizeof(int) * num_fact));
  CubDebugExit(g_allocator.DeviceAllocate((void**)&d_fact_val, sizeof(int) * num_fact));

  int *h_dim_key = NULL;
  int *h_dim_val = NULL;
  int *h_fact_fkey = NULL;
  int *h_fact_val = NULL;

  create_relation_pk(h_dim_key, h_dim_val, num_dim);
  create_relation_fk(h_fact_fkey, h_fact_val, num_fact, num_dim);

  CubDebugExit(hipMemcpy(d_dim_key, h_dim_key, sizeof(int) * num_dim, hipMemcpyHostToDevice));
  CubDebugExit(hipMemcpy(d_dim_val, h_dim_val, sizeof(int) * num_dim, hipMemcpyHostToDevice));
  CubDebugExit(hipMemcpy(d_fact_fkey, h_fact_fkey, sizeof(int) * num_fact, hipMemcpyHostToDevice));
  CubDebugExit(hipMemcpy(d_fact_val, h_fact_val, sizeof(int) * num_fact, hipMemcpyHostToDevice));

  /*cout << "DIM TABLE:" << endl;
  for (int i = 0; i < num_dim; i++) cout << h_dim_key[i] << "..." << h_dim_val[i] << endl;
  cout << endl;

  cout << "FACT TABLE:" << endl;
  for (int i = 0; i < num_fact; i++) cout << h_fact_fkey[i] << "..." << h_fact_val[i] << endl;
  cout << endl;*/

  for (int j = 0; j < num_trials; j++) {
    cout << "TRIAL " << j << endl;
    TimeKeeper t = hashJoin(d_dim_key, d_dim_val, d_fact_fkey, d_fact_val, num_dim, num_fact, g_allocator);
    cout<< "{"
      << "\"num_dim\":" << num_dim
      << ",\"num_fact\":" << num_fact
      << ",\"radix\":" << 0
      << ",\"time_partition_build\":" << 0
      << ",\"time_partition_probe\":" << 0
      << ",\"time_partition_total\":" << 0
      << ",\"time_build\":" << t.time_build
      << ",\"time_probe\":" << t.time_probe
      << ",\"time_extra\":" << t.time_extra
      << ",\"time_join_total\":" << t.time_total
      << "}" << endl;
    cout << endl;
  }

  int *d_fact_fkey_copy;
  CubDebugExit(g_allocator.DeviceAllocate((void**)&d_fact_fkey_copy, sizeof(int) * num_fact));
  CubDebugExit(hipMemcpy(d_fact_fkey_copy, d_fact_fkey, sizeof(int) * num_fact, hipMemcpyDeviceToDevice));

  int *d_fact_val_copy;
  CubDebugExit(g_allocator.DeviceAllocate((void**)&d_fact_val_copy, sizeof(int) * num_fact));
  CubDebugExit(hipMemcpy(d_fact_val_copy, d_fact_val, sizeof(int) * num_fact, hipMemcpyDeviceToDevice));

  int *d_buffer1;
  int *d_buffer2;
  CubDebugExit(g_allocator.DeviceAllocate((void**)&d_buffer1, sizeof(int) * num_fact));
  CubDebugExit(g_allocator.DeviceAllocate((void**)&d_buffer2, sizeof(int) * num_fact));

  // Checking against hash join on CPU
  int num_slots = num_dim;
  int *hash_table = new int[num_slots * 2];
  long long check_res = 0;
  RunHashJoinCPU(h_dim_key, h_dim_val, h_fact_fkey, h_fact_val, hash_table, &check_res, num_dim, num_fact, num_slots);
  cout << "CPU answer: " << check_res << endl; 

  CLEANUP(d_dim_key);
  CLEANUP(d_dim_val);
  CLEANUP(d_fact_fkey);
  CLEANUP(d_fact_val);

  return 0;
}

