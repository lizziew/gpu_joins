#include "hip/hip_runtime.h"
// make setup; make gpu_join; ./bin/gpu/join > out
// Ensure printing of CUDA runtime errors to console
#define HIPCUB_STDERR

#include <iostream>
#include <stdio.h>
#include <hiprand.h>

#include <hip/hip_runtime.h>
#include <cub/util_allocator.cuh>
#include <cub/device/device_scan.cuh>
#include <cub/device/device_radix_sort.cuh>
#include "cub/test/test_util.h"

#include "utils/generator.h"
#include "utils/gpu_utils.h"

#include <thrust/copy.h>

using namespace std;
using namespace hipcub;

#define DEBUG 1
#define NGPU 2

__device__ __forceinline__
int HASH(const int key, const int num_slots) {
  return key & (num_slots - 1);
}

__forceinline__
int HHASH(const int key, const int num_slots) {
  return key & (num_slots - 1);
}

void partition_dev(int* h_key, int* h_val, int** key_partitions, int** val_partitions, int num_slots, int count[NGPU]) {
  memset(count, 0, sizeof(count)); 

  for (int i = 0; i < num_slots; i ++) {
    int key = h_key[i];
    int hash = (HHASH(key, num_slots) % NGPU);

    printf("Assigning %d to partition %d at index %d\n", key, hash, count[hash]);
    key_partitions[hash][count[hash]] = key;
    val_partitions[hash][count[hash]] = h_val[i];

    count[hash]++; 
  }
}

__global__
void build_hashtable_dev(int *d_dim_key, int *d_dim_val, int num_tuples, int *hash_table, int num_slots) {
  int offset = blockIdx.x * blockDim.x + threadIdx.x;

  int key = d_dim_key[offset];
  int val = d_dim_val[offset];
  int hash = HASH(key, num_slots);

  hash_table[hash << 1] = key;
  hash_table[(hash << 1) + 1] = val;
}

__global__
void probe_hashtable_dev(int *d_fact_fkey, int *d_fact_val, int num_tuples, int *hash_table, int num_slots, unsigned long long *res) {
  int offset = blockIdx.x*blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;

  unsigned long long checksum = 0;

  for (int i = offset; i < num_tuples; i += stride) {
    int key = d_fact_fkey[i];
    int val = d_fact_val[i];
    int hash = HASH(key, num_slots);

    int2 slot = reinterpret_cast<int2*>(hash_table)[hash];
    if (slot.x == key) {
      checksum += slot.y + val;
    }
  }

  atomicAdd(res, checksum);
}

struct TimeKeeper {
  float time_build;
  float time_probe;
  float time_extra;
  float time_total;
};

static int num_runs = 0;
static unsigned long long* h_res = 0;

TimeKeeper hashJoin(int* h_dim_key, int* h_dim_val, int* h_fact_fkey, int* h_fact_val, int* d_dim_key, int* d_dim_val, int* d_fact_fkey, int* d_fact_val, int num_dim, int num_fact, CachingDeviceAllocator&  g_allocator) {
  SETUP_TIMING();

  // Partition
  int** h_dim_key_partitions = new int*[NGPU];
  for(int i = 0; i < NGPU; ++i) h_dim_key_partitions[i] = new int[num_dim];
  int** h_dim_val_partitions = new int*[NGPU];
  for(int i = 0; i < NGPU; ++i) h_dim_val_partitions[i] = new int[num_dim];
  int** h_fact_key_partitions = new int*[NGPU];
  for(int i = 0; i < NGPU; ++i) h_fact_key_partitions[i] = new int[num_fact];
  int** h_fact_val_partitions = new int*[NGPU];
  for(int i = 0; i < NGPU; ++i) h_fact_val_partitions[i] = new int[num_fact];
  int* h_dim_count; 
  int* h_fact_count;

  printf("Partitioning dim...\n");
  partition_dev(h_dim_key, h_dim_val, h_dim_key_partitions, h_dim_val_partitions, num_dim, h_dim_count); 
  for (int i = 0; i < NGPU; i++) {
    for (int j = 0; j < h_dim_count[i]; i++) {
      printf("%d:%d ", h_dim_key_partitions[i][j], h_dim_val_partitions[i][j]);
    }
    printf("\n");
  } 

  printf("Partitioning fact...\n");
  partition_dev(h_fact_fkey, h_fact_val, h_fact_key_partitions, h_fact_val_partitions, num_fact, h_fact_count);
  for (int i = 0; i < NGPU; i++) {
    for (int j = 0; j < h_fact_count[i]; i++) {
      printf("%d ", h_fact_key_partitions[i][j], h_fact_val_partitions[i][j]);
    }
    printf("\n");
  } 

  // Build hashtable
  printf("Building hashtables...\n");
  int* hash_table = NULL;
  unsigned long long* res;
  int num_slots = num_dim;
  float time_build, time_probe, time_memset, time_memset2;

  int* d_dim_key_partitions;
  int* d_dim_val_partitions; 

  ALLOCATE(hash_table, sizeof(int) * 2 * num_dim);
  ALLOCATE(res, sizeof(long long));

  ALLOCATE(d_dim_key_partitions, sizeof(int) * NGPU * num_dim);
  ALLOCATE(d_dim_val_partitions, sizeof(int) * NGPU * num_dim); 

  /*
  CubDebugExit(hipMemcpy(d_dim_key_partitions, h_dim_key_partitions, sizeof(int) * NGPU * num_dim, hipMemcpyHostToDevice));
  CubDebugExit(hipMemcpy(d_dim_val_partitions, h_dim_val_partitions, sizeof(int) * NGPU * num_dim, hipMemcpyHostToDevice));

  TIME_FUNC(hipMemset(hash_table, 0, num_slots * sizeof(int) * 2), time_memset);
  TIME_FUNC(hipMemset(res, 0, sizeof(long long)), time_memset2);

  // num_dim/128
  TIME_FUNC((build_hashtable_dev<<<128, 128>>>(d_dim_key_partitions, d_dim_val_partitions, num_dim, hash_table, num_slots)), time_build);
  hipDeviceSynchronize(); 

  // Probe hashtable
  TIME_FUNC((probe_hashtable_dev<<<192, 256>>>(d_fact_fkey, d_fact_val, num_fact, hash_table, num_slots, res)), time_probe);
  hipDeviceSynchronize(); 

#if DEBUG
  cout << "{" << "\"time_memset\":" << time_memset
    << ",\"time_build\"" << time_build
    << ",\"time_probe\":" << time_probe << "}" << endl;
#endif

  num_runs += 1;
  if (num_runs == 3) {
    h_res = new unsigned long long[1];
    CubDebugExit(hipMemcpy(h_res, res, sizeof(long long), hipMemcpyDeviceToHost));
    cout << h_res[0] << endl;
  }

  CLEANUP(hash_table);
  CLEANUP(res);
  */ 
  TimeKeeper t = {time_build, time_probe, time_memset, time_build + time_probe + time_memset};
  return t;
}

void RunHashJoinCPU(int *dim_key, int* dim_val, int* fact_fkey, int* fact_val, int* hash_table, long long* res, int num_dim, int num_fact, int num_slots) {
  for (int i = 0; i < num_dim; i++) {
    int key = dim_key[i];
    int val = dim_val[i];

    int hash = key & (num_slots - 1);

    hash_table[hash << 1] = key;
    hash_table[(hash << 1) + 1] = val;
  }

  for (int i = 0; i < num_fact; i++) {
    int key = fact_fkey[i];
    int val = fact_val[i];

    int hash = key & (num_slots - 1);

    if (hash_table[hash << 1] == key) {
      *res = *res + (hash_table[(hash << 1) + 1] + val);
    } 
  }
}

//---------------------------------------------------------------------
// Globals, constants and typedefs
//---------------------------------------------------------------------

bool                    g_verbose = false;  // Whether to display input/output to console
CachingDeviceAllocator  g_allocator(true);  // Caching allocator for device memory


#define CLEANUP(vec) if(vec)CubDebugExit(g_allocator.DeviceFree(vec))

//---------------------------------------------------------------------
// Main
//---------------------------------------------------------------------
int main(int argc, char** argv)
{
  int num_fact           = 16; // 256 * 1 << 20 , 1 << 28
  int num_dim            = 4; // 16 * 1 << 20 , 1 << 16
  int num_trials         = 3;

  // Initialize command line
  CommandLineArgs args(argc, argv);
  args.GetCmdLineArgument("n", num_fact);
  args.GetCmdLineArgument("d", num_dim);
  args.GetCmdLineArgument("t", num_trials);

  // Print usage
  if (args.CheckCmdLineFlag("help"))
  {
    printf("%s "
        "[--n=<num fact>] "
        "[--d=<num dim>] "
        "[--t=<num trials>] "
        "[--device=<device-id>] "
        "[--v] "
        "\n", argv[0]);
    exit(0);
  }

  int log2 = 0;
  int num_dim_dup = num_dim >> 1;
  while (num_dim_dup) {
    num_dim_dup >>= 1;
    log2 += 1;
  }

  // Initialize device
  CubDebugExit(args.DeviceInit());

  // Allocate problem device arrays
  int *d_dim_key = NULL;
  int *d_dim_val = NULL;
  int *d_fact_fkey = NULL;
  int *d_fact_val = NULL;
  CubDebugExit(g_allocator.DeviceAllocate((void**)&d_dim_key, sizeof(int) * num_dim));
  CubDebugExit(g_allocator.DeviceAllocate((void**)&d_dim_val, sizeof(int) * num_dim));
  CubDebugExit(g_allocator.DeviceAllocate((void**)&d_fact_fkey, sizeof(int) * num_fact));
  CubDebugExit(g_allocator.DeviceAllocate((void**)&d_fact_val, sizeof(int) * num_fact));

  int *h_dim_key = NULL;
  int *h_dim_val = NULL;
  int *h_fact_fkey = NULL;
  int *h_fact_val = NULL;

  create_relation_pk(h_dim_key, h_dim_val, num_dim);
  create_relation_fk(h_fact_fkey, h_fact_val, num_fact, num_dim);

  CubDebugExit(hipMemcpy(d_dim_key, h_dim_key, sizeof(int) * num_dim, hipMemcpyHostToDevice));
  CubDebugExit(hipMemcpy(d_dim_val, h_dim_val, sizeof(int) * num_dim, hipMemcpyHostToDevice));
  CubDebugExit(hipMemcpy(d_fact_fkey, h_fact_fkey, sizeof(int) * num_fact, hipMemcpyHostToDevice));
  CubDebugExit(hipMemcpy(d_fact_val, h_fact_val, sizeof(int) * num_fact, hipMemcpyHostToDevice));

  cout << "DIM TABLE:" << endl;
  for (int i = 0; i < num_dim; i++) cout << h_dim_key[i] << "..." << h_dim_val[i] << endl;
  cout << endl;

  cout << "FACT TABLE:" << endl;
  for (int i = 0; i < num_fact; i++) cout << h_fact_fkey[i] << "..." << h_fact_val[i] << endl;
  cout << endl;

  for (int j = 0; j < num_trials; j++) {
    cout << "TRIAL " << j << endl;
    TimeKeeper t = hashJoin(h_dim_key, h_dim_val, h_fact_fkey, h_fact_val, d_dim_key, d_dim_val, d_fact_fkey, d_fact_val, num_dim, num_fact, g_allocator);
    cout<< "{"
      << "\"num_dim\":" << num_dim
      << ",\"num_fact\":" << num_fact
      << ",\"radix\":" << 0
      << ",\"time_partition_build\":" << 0
      << ",\"time_partition_probe\":" << 0
      << ",\"time_partition_total\":" << 0
      << ",\"time_build\":" << t.time_build
      << ",\"time_probe\":" << t.time_probe
      << ",\"time_extra\":" << t.time_extra
      << ",\"time_join_total\":" << t.time_total
      << "}" << endl;
    cout << endl;
  }

  int *d_fact_fkey_copy;
  CubDebugExit(g_allocator.DeviceAllocate((void**)&d_fact_fkey_copy, sizeof(int) * num_fact));
  CubDebugExit(hipMemcpy(d_fact_fkey_copy, d_fact_fkey, sizeof(int) * num_fact, hipMemcpyDeviceToDevice));

  int *d_fact_val_copy;
  CubDebugExit(g_allocator.DeviceAllocate((void**)&d_fact_val_copy, sizeof(int) * num_fact));
  CubDebugExit(hipMemcpy(d_fact_val_copy, d_fact_val, sizeof(int) * num_fact, hipMemcpyDeviceToDevice));

  int *d_buffer1;
  int *d_buffer2;
  CubDebugExit(g_allocator.DeviceAllocate((void**)&d_buffer1, sizeof(int) * num_fact));
  CubDebugExit(g_allocator.DeviceAllocate((void**)&d_buffer2, sizeof(int) * num_fact));

  // Checking against hash join on CPU
  int num_slots = num_dim;
  int *hash_table = new int[num_slots * 2];
  long long check_res = 0;
  RunHashJoinCPU(h_dim_key, h_dim_val, h_fact_fkey, h_fact_val, hash_table, &check_res, num_dim, num_fact, num_slots);
  cout << "CPU answer: " << check_res << endl; 

  CLEANUP(d_dim_key);
  CLEANUP(d_dim_val);
  CLEANUP(d_fact_fkey);
  CLEANUP(d_fact_val);

  return 0;
}

